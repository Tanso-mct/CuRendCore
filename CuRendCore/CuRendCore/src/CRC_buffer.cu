﻿#include "CRC_pch.h"
#include "CRC_funcs.cuh"

#include "CRC_buffer.cuh"

std::unique_ptr<ICRCContainable> CRCBufferFactoryL0_0::Create(IDESC &desc) const
{
    CRC_BUFFER_DESC* bufferDesc = CRC::As<CRC_BUFFER_DESC>(&desc);
    if (!bufferDesc)
    {
#ifndef NDEBUG
        CRC::CoutWarning("Failed to create buffer from desc. Desc is not CRC_BUFFER_DESC.");
#endif
        return nullptr;
    }

    std::unique_ptr<CRCBuffer> buffer = std::make_unique<CRCBuffer>(*bufferDesc);
    return buffer;
}

CRCBuffer::CRCBuffer(CRC_BUFFER_DESC &desc)
{
    D3D11_BUFFER_DESC& src = desc.desc_;
    desc_ = src;

    Malloc(src.ByteWidth);
    if (desc.initialData_.pSysMem)
    {
        CRC::CheckCuda(hipMemcpy
        (
            memPtr_, desc.initialData_.pSysMem, byteWidth_, hipMemcpyHostToDevice
        ));
    }
}

CRCBuffer::~CRCBuffer()
{
    if (memPtr_) Free();
}

HRESULT CRCBuffer::GetType(UINT& rcType)
{
    rcType = rcType_;
    return S_OK;
}

const void CRCBuffer::GetDesc(D3D11_BUFFER_DESC *dst)
{
    std::memcpy(dst, &desc_, sizeof(D3D11_BUFFER_DESC));
}

void CRCBuffer::Malloc(UINT byteWidth)
{
    if (memPtr_)
    {
#ifndef NDEBUG
        CRC::CoutError("Buffer device memory already allocated.");
#endif
        throw std::runtime_error("Buffer device memory already allocated.");
    }

    byteWidth_ = byteWidth;
    CRC::CheckCuda(hipMalloc(&memPtr_, byteWidth_));

#ifndef NDEBUG
    CRC::Cout
    (
        "Buffer device memory allocated.", "\n",
        "ByteWidth :", byteWidth_
    );
#endif
}

void CRCBuffer::Free()
{
    if (!memPtr_)
    {
#ifndef NDEBUG
        CRC::CoutError("Buffer device memory not allocated.");
#endif
        throw std::runtime_error("Buffer device memory not allocated.");
    }

    byteWidth_ = 0;

    CRC::CheckCuda(hipFree(memPtr_));
    memPtr_ = nullptr;

#ifndef NDEBUG
    CRC::Cout("Buffer device memory free.");
#endif
}

std::unique_ptr<ICRCContainable> CRCID3D11BufferFactoryL0_0::Create(IDESC &desc) const
{
    CRC_BUFFER_DESC* bufferDesc = CRC::As<CRC_BUFFER_DESC>(&desc);
    if (!bufferDesc)
    {
#ifndef NDEBUG
        CRC::CoutWarning("Failed to create buffer from desc. Desc is not CRC_BUFFER_DESC.");
#endif
        return nullptr;
    }

    if (!bufferDesc->d3d11Device_)
    {
#ifndef NDEBUG
        CRC::CoutWarning("Failed to create buffer. D3D11 device is nullptr.");
#endif
        return nullptr;
    }

    std::unique_ptr<CRCID3D11Buffer> buffer = std::make_unique<CRCID3D11Buffer>();

    HRESULT hr = bufferDesc->d3d11Device_->CreateBuffer
    (
        &bufferDesc->desc_, &bufferDesc->initialData_, buffer->Get().GetAddressOf()
    );
    if (FAILED(hr))
    {
#ifndef NDEBUG
        CRC::CoutError("Failed to create buffer.");
#endif
        throw std::runtime_error("Failed to create buffer.");
    }

    return buffer;
}

const void CRCID3D11Buffer::GetDesc(D3D11_BUFFER_DESC *dst)
{
    d3d11Buffer_->GetDesc(dst);
}

HRESULT CRCID3D11Buffer::GetType(UINT& rcType)
{
    rcType = 0;
    return S_OK;
}