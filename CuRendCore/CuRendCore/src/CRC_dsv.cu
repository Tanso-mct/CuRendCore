﻿#include "CRC_pch.h"
#include "CRC_funcs.cuh"

#include "CRC_dsv.cuh"

std::unique_ptr<ICRCResource> &CRCDepthStencilView::GetResource()
{
    return resource;
}

std::unique_ptr<ICRCResource> &CRCID3D11DepthStencilView::GetResource()
{
    return emptyResource;
}
