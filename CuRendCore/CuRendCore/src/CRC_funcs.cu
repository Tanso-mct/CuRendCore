#include "hip/hip_runtime.h"
﻿#include "CRC_pch.h"
#include "CRC_funcs.cuh"

#include "CRC_window.h"
#include "CRC_scene.h"

#include "CRC_container.h"
#include "CRC_event.h"

#include "CRC_device.cuh"
#include "CRC_swap_chain.cuh"

#include "CRC_texture.cuh"

HRESULT CRC::ShowWindowCRC(HWND& hWnd)
{
    if (!hWnd) return E_FAIL;

    ShowWindow(hWnd, SW_SHOW);
    UpdateWindow(hWnd);

    return S_OK;
}

CRC_API HRESULT CRC::CreateD3D11DeviceAndSwapChain
(
    CRC_SWAP_CHAIN_DESC& desc,
    Microsoft::WRL::ComPtr<ID3D11Device> &device, Microsoft::WRL::ComPtr<IDXGISwapChain> &swapChain
){
    UINT createDeviceFlags = 0;
    D3D_FEATURE_LEVEL featureLevel;
    const D3D_FEATURE_LEVEL featureLevelArray[2] = { D3D_FEATURE_LEVEL_11_0, D3D_FEATURE_LEVEL_10_0, };

    HRESULT hr = D3D11CreateDeviceAndSwapChain
    (
        nullptr, D3D_DRIVER_TYPE_HARDWARE, nullptr, createDeviceFlags, featureLevelArray, 2, 
        D3D11_SDK_VERSION, &desc.GetDxgiDesc(), &swapChain, &device, &featureLevel, nullptr
    );

    if (hr == DXGI_ERROR_UNSUPPORTED) // Try high-performance WARP software driver if hardware is not available.
    {
        hr = D3D11CreateDeviceAndSwapChain
        (
            nullptr, D3D_DRIVER_TYPE_WARP, nullptr, createDeviceFlags, featureLevelArray, 2, 
            D3D11_SDK_VERSION, &desc.GetDxgiDesc(), &swapChain, &device, &featureLevel, nullptr
        );
    }

    return hr;
}

CRC_API HRESULT CRC::CreateCRCDeviceAndSwapChain
(
    CRC_DEVICE_DESC &deviceDesc, CRC_SWAP_CHAIN_DESC &swapChainDesc, 
    const ICRCFactory &deviceFactory, const ICRCFactory &swapChainFactory, 
    std::unique_ptr<ICRCContainable> &device, std::unique_ptr<ICRCContainable> &swapChain
){
    device = deviceFactory.Create(deviceDesc);
    if (!device) return E_FAIL;

    swapChain = swapChainFactory.Create(swapChainDesc);
    if (!swapChain) return E_FAIL;

    return S_OK;
}

UINT CRC::GetBytesPerPixel(const DXGI_FORMAT &format)
{
    switch (format)
    {
    case DXGI_FORMAT_R8G8B8A8_UNORM:
    case DXGI_FORMAT_D24_UNORM_S8_UINT:
        return 4;

    default:
        throw std::runtime_error("This DXGI_FORMAT is not supported by CuRendCore.");
        return 0;
    }
}

HRESULT CRC::CreateCudaChannelDescFromDXGIFormat(hipChannelFormatDesc &channelDesc, const DXGI_FORMAT &format)
{
    switch (format)
    {
    case DXGI_FORMAT_R8G8B8A8_UNORM:
    case DXGI_FORMAT_D24_UNORM_S8_UINT:
        channelDesc = hipCreateChannelDesc<uchar4>();
        return S_OK;

    default:
        throw std::runtime_error("This DXGI_FORMAT is not supported by CuRendCore.");
        return E_FAIL;
    }
}

CRC_API void CRC::CheckCuda(hipError_t call)
{
    if (call != hipSuccess)
    {
        std::string err = "[CUDA ERROR] Code: " + std::to_string(call) + ", Reason: " + hipGetErrorString(call);
#ifndef NDEBUG
        CoutError(err);
#endif
        throw std::runtime_error(err);
    }
}

HRESULT CRC::RegisterCudaResources
(
    std::vector<hipGraphicsResource_t> &cudaResources, const hipGraphicsRegisterFlags &flags, 
    const UINT &bufferCount, IDXGISwapChain *d3d11SwapChain
){
    cudaResources.resize(bufferCount);
    std::vector<ID3D11Texture2D*> buffers(bufferCount);

    DXGI_SWAP_CHAIN_DESC desc;
    d3d11SwapChain->GetDesc(&desc);
    UINT bufferCountFromDesc = desc.BufferCount;

    for (UINT i = 0; i < bufferCountFromDesc; i++)
    {
        HRESULT hr = d3d11SwapChain->GetBuffer(i, __uuidof(ID3D11Texture2D), (void**)&buffers[i]);
        if (FAILED(hr))
        {
#ifndef NDEBUG
            CoutError("Failed to get buffers from DXGI swap chain.");
#endif
            return E_FAIL;
        }

        hipError_t err = cudaGraphicsD3D11RegisterResource
        (
            &cudaResources[i], buffers[i], flags
        );
        if (err != hipSuccess)
        {
#ifndef NDEBUG
            CoutError("Failed to register CUDA resources.");
#endif
            return E_FAIL;
        }
    }

    for (int i = 0; i < bufferCountFromDesc; i++)
    {
        buffers[i]->Release();
    }

#ifndef NDEBUG
    Cout("Registered CUDA resources.");
#endif

    return S_OK;
}

HRESULT CRC::RegisterCudaResource
(
    hipGraphicsResource_t &cudaResource, const hipGraphicsRegisterFlags &flags, 
    ID3D11Texture2D *d3d11Texture
){
    hipError_t err = cudaGraphicsD3D11RegisterResource(&cudaResource, d3d11Texture, flags);
    if (err != hipSuccess)
    {
#ifndef NDEBUG
        CoutError("Failed to register CUDA resource.");
#endif
        return E_FAIL;
    }

#ifndef NDEBUG
    Cout("Registered CUDA resource.");
#endif

    return S_OK;
}

HRESULT CRC::UnregisterCudaResources(std::vector<hipGraphicsResource_t> &cudaResources)
{
    for (int i = 0; i < cudaResources.size(); ++i) 
    {
        hipError_t err = hipGraphicsUnregisterResource(cudaResources[i]);
        if (err != hipSuccess)
        {
#ifndef NDEBUG
            CoutError("Failed to unregister CUDA resources.");
#endif
            return E_FAIL;
        }
    }

#ifndef NDEBUG
    Cout("Unregistered CUDA resources.");
#endif

    return S_OK;
}

HRESULT CRC::UnregisterCudaResource(hipGraphicsResource_t &cudaResource)
{
    hipError_t err = hipGraphicsUnregisterResource(cudaResource);
    if (err != hipSuccess)
    {
#ifndef NDEBUG
        CoutError("Failed to unregister CUDA resource.");
#endif
        return E_FAIL;
    }

#ifndef NDEBUG
    Cout("Unregistered CUDA resource.");
#endif

    return S_OK;
}

HRESULT CRC::UnregisterCudaResourcesAtSwapChain
(
    std::vector<hipGraphicsResource_t> &cudaResources, 
    Microsoft::WRL::ComPtr<IDXGISwapChain> &d3d11SwapChain, UINT &frameIndex, const UINT& bufferCount
){
    for (int i = 0; i < cudaResources.size(); ++i) 
    {
        if (i == frameIndex) continue;

        hipError_t err = hipGraphicsUnregisterResource(cudaResources[i]);
        if (err != hipSuccess)
        {
#ifndef NDEBUG
            CoutError("Failed to unregister CUDA resources in swap chain.");
#endif
            return E_FAIL;
        }
    }

    d3d11SwapChain->Present(0, 0);
    hipError_t err = hipGraphicsUnregisterResource(cudaResources[frameIndex]);
    if (err != hipSuccess)
    {
#ifndef NDEBUG
        CoutError("Failed to unregister CUDA resources in swap chain.");
#endif
        return E_FAIL;
    }

    frameIndex = (frameIndex + 1) % bufferCount;

#ifndef NDEBUG
    Cout("Unregistered CUDA resources in swap chain.");
#endif

    return S_OK;
}

HRESULT CRC::MapCudaResource(hipGraphicsResource_t& cudaResource, hipStream_t stream)
{
    hipError_t err = hipGraphicsMapResources(1, &cudaResource, stream);
    if (err != hipSuccess)
    {
#ifndef NDEBUG
        CoutError("Failed to map CUDA resource.");
#endif
        return E_FAIL;
    }

#ifndef NDEBUG
    Cout("Mapped CUDA resource.");
#endif

    return S_OK;
}

HRESULT CRC::UnmapCudaResource(hipGraphicsResource_t& cudaResource, hipStream_t stream)
{
    hipError_t err = hipGraphicsUnmapResources(1, &cudaResource, stream);
    if (err != hipSuccess)
    {
#ifndef NDEBUG
        CoutError("Failed to unmap CUDA resource.");
#endif
        return E_FAIL;
    }

#ifndef NDEBUG
    Cout("Unmapped CUDA resource.");
#endif

    return S_OK;
}

hipArray_t CRC::GetCudaMappedArray(hipGraphicsResource_t& cudaResource)
{
    hipArray_t hipArray;
    hipError_t err = hipGraphicsSubResourceGetMappedArray(&hipArray, cudaResource, 0, 0);
    if (err != hipSuccess)
    {
#ifndef NDEBUG
        CoutError("Failed to get CUDA mapped array.");
#endif
        return nullptr;
    }

    return hipArray;
}

CRC_API std::unique_ptr<ICRCTexture2D> CRC::CreateCudaSurfaceObjects
(
    hipGraphicsResource_t& cudaResource, const UINT& width, const UINT& height, const DXGI_FORMAT& format
){
    HRESULT hr = CRC::MapCudaResource(cudaResource);
    if (FAILED(hr))
    {
#ifndef NDEBUG
        CRC::CoutError("Failed to create surface objects by mapping CUDA resources.");
#endif
        return nullptr;
    }

    hipArray_t backBufferArray = CRC::GetCudaMappedArray(cudaResource);
    if (!backBufferArray)
    {
#ifndef NDEBUG
        CRC::CoutError("Failed to create surface objects by getting CUDA mapped array.");
#endif
        return nullptr;
    }

    std::unique_ptr<ICRCTexture2D> rtTexture = std::make_unique<CRCTextureSurface>
    (
        CRC::GetBytesPerPixel(format) * width * height, 
        CRC::GetBytesPerPixel(format) * width, 
        CRC::GetBytesPerPixel(format) * width * height
    );

    CRCTextureSurface* backSurface = CRC::As<CRCTextureSurface>(rtTexture.get());
    if (!backSurface)
    {
#ifndef NDEBUG
        CRC::CoutError("Failed to create surface objects by casting back surface to ICRCMem.");
#endif
        return nullptr;
    }

    hipResourceDesc resDesc = {};
    resDesc.resType = hipResourceTypeArray;
    resDesc.res.array.array = backBufferArray;
    hr = CRC::CreateCudaSurfaceObject(backSurface->GetSurfaceObj(), resDesc);

    hr = CRC::UnmapCudaResource(cudaResource);
    if (FAILED(hr))
    {
#ifndef NDEBUG
        CRC::CoutError("Failed to create surface objects by unmapping CUDA resources.");
#endif
        return nullptr;
    }

    return rtTexture;
}

HRESULT CRC::CreateCudaSurfaceObject(hipSurfaceObject_t &surfaceObject, const hipResourceDesc &desc)
{
    hipError_t err = hipCreateSurfaceObject(&surfaceObject, &desc);
    if (err != hipSuccess)
    {
#ifndef NDEBUG
        CoutError("Failed to create CUDA surface object.");
#endif
        return E_FAIL;
    }

#ifndef NDEBUG
    Cout("Created CUDA surface object.");
#endif

    return S_OK;
}
