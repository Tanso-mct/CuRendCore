﻿#include "CRC_pch.h"
#include "CRC_funcs.cuh"

#include "CRC_window.h"
#include "CRC_scene.h"

#include "CRC_container.h"
#include "CRC_event.h"

#include "CRC_device.cuh"
#include "CRC_swap_chain.cuh"

HRESULT CRC::ShowWindowCRC(HWND& hWnd)
{
    if (!hWnd) return E_FAIL;

    ShowWindow(hWnd, SW_SHOW);
    UpdateWindow(hWnd);

    return S_OK;
}

CRC_API HRESULT CRC::CreateD3D11DeviceAndSwapChain
(
    CRC_SWAP_CHAIN_DESC& desc,
    Microsoft::WRL::ComPtr<ID3D11Device> &device, Microsoft::WRL::ComPtr<IDXGISwapChain> &swapChain
){
    UINT createDeviceFlags = 0;
    D3D_FEATURE_LEVEL featureLevel;
    const D3D_FEATURE_LEVEL featureLevelArray[2] = { D3D_FEATURE_LEVEL_11_0, D3D_FEATURE_LEVEL_10_0, };

    HRESULT hr = D3D11CreateDeviceAndSwapChain
    (
        nullptr, D3D_DRIVER_TYPE_HARDWARE, nullptr, createDeviceFlags, featureLevelArray, 2, 
        D3D11_SDK_VERSION, &desc.GetDxgiDesc(), &swapChain, &device, &featureLevel, nullptr
    );

    if (hr == DXGI_ERROR_UNSUPPORTED) // Try high-performance WARP software driver if hardware is not available.
    {
        hr = D3D11CreateDeviceAndSwapChain
        (
            nullptr, D3D_DRIVER_TYPE_WARP, nullptr, createDeviceFlags, featureLevelArray, 2, 
            D3D11_SDK_VERSION, &desc.GetDxgiDesc(), &swapChain, &device, &featureLevel, nullptr
        );
    }

    return hr;
}

UINT CRC::GetBytesPerPixel(const DXGI_FORMAT &format)
{
    switch (format)
    {
    case DXGI_FORMAT_R8G8B8A8_UNORM:
    case DXGI_FORMAT_D24_UNORM_S8_UINT:
        return 4;

    default:
        throw std::runtime_error("This DXGI_FORMAT is not supported by CuRendCore.");
    }
}

CRC_API void CRC::CheckCuda(hipError_t call)
{
    if (call != hipSuccess)
    {
        std::string err = "[CUDA ERROR] Code: " + std::to_string(call) + ", Reason: " + hipGetErrorString(call);
#ifndef NDEBUG
        CoutError(err);
#endif
        throw std::runtime_error(err);
    }
}

HRESULT CRC::RegisterCudaResources
(
    std::vector<hipGraphicsResource_t> &cudaResources, const hipGraphicsRegisterFlags &flags, 
    const UINT &bufferCount, IDXGISwapChain *d3d11SwapChain
){
    cudaResources.resize(bufferCount);
    std::vector<ID3D11Texture2D*> buffers(bufferCount);

    DXGI_SWAP_CHAIN_DESC desc;
    d3d11SwapChain->GetDesc(&desc);
    UINT bufferCountFromDesc = desc.BufferCount;

    for (UINT i = 0; i < bufferCountFromDesc; i++)
    {
        HRESULT hr = d3d11SwapChain->GetBuffer(i, __uuidof(ID3D11Texture2D), (void**)&buffers[i]);
        if (FAILED(hr)) return hr;

        hipError_t err = cudaGraphicsD3D11RegisterResource
        (
            &cudaResources[i], buffers[i], flags
        );
        if (err != hipSuccess) return E_FAIL;
    }

    for (int i = 0; i < bufferCountFromDesc; i++)
    {
        buffers[i]->Release();
    }

    return S_OK;
}

HRESULT CRC::RegisterCudaResource
(
    hipGraphicsResource_t &cudaResource, const hipGraphicsRegisterFlags &flags, 
    ID3D11Texture2D *d3d11Texture
){
    hipError_t err = cudaGraphicsD3D11RegisterResource(&cudaResource, d3d11Texture, flags);
    if (err != hipSuccess) return E_FAIL;

    return S_OK;
}

HRESULT CRC::UnregisterCudaResources(std::vector<hipGraphicsResource_t> &cudaResources)
{
    for (int i = 0; i < cudaResources.size(); ++i) 
    {
        hipError_t err = hipGraphicsUnregisterResource(cudaResources[i]);
        if (err != hipSuccess) return E_FAIL;
    }

    return S_OK;
}

HRESULT CRC::UnregisterCudaResource(hipGraphicsResource_t &cudaResource)
{
    hipError_t err = hipGraphicsUnregisterResource(cudaResource);
    if (err != hipSuccess) return E_FAIL;

    return S_OK;
}

HRESULT CRC::UnregisterCudaResourcesAtSwapChain
(
    std::vector<hipGraphicsResource_t> &cudaResources, 
    Microsoft::WRL::ComPtr<IDXGISwapChain> &d3d11SwapChain, UINT &frameIndex, const UINT& bufferCount
){
    for (int i = 0; i < cudaResources.size(); ++i) 
    {
        if (i == frameIndex) continue;

        hipError_t err = hipGraphicsUnregisterResource(cudaResources[i]);
        if (err != hipSuccess) return E_FAIL;
    }

    d3d11SwapChain->Present(0, 0);
    hipError_t err = hipGraphicsUnregisterResource(cudaResources[frameIndex]);
    if (err != hipSuccess) return E_FAIL;

    frameIndex = (frameIndex + 1) % bufferCount;

    return S_OK;
}

HRESULT CRC::MapCudaResource(hipGraphicsResource_t& cudaResource, hipStream_t stream)
{
    hipError_t err = hipGraphicsMapResources(1, &cudaResource, stream);
    if (err != hipSuccess) return E_FAIL;

    return S_OK;
}

HRESULT CRC::UnmapCudaResource(hipGraphicsResource_t& cudaResource, hipStream_t stream)
{
    hipError_t err = hipGraphicsUnmapResources(1, &cudaResource, stream);
    if (err != hipSuccess) return E_FAIL;

    return S_OK;
}

hipArray_t CRC::GetCudaMappedArray(hipGraphicsResource_t& cudaResource)
{
    hipArray_t hipArray;
    hipError_t err = hipGraphicsSubResourceGetMappedArray(&hipArray, cudaResource, 0, 0);
    if (err != hipSuccess) return nullptr;

    return hipArray;
}
