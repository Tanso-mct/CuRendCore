﻿#include "CRC_pch.h"
#include "CRC_funcs.cuh"

#include "CRC_window.h"
#include "CRC_scene.h"

#include "CRC_container.h"
#include "CRC_event.h"

#include "CRC_device.cuh"
#include "CRC_swap_chain.cuh"

HRESULT CRC::ShowWindowCRC(HWND& hWnd)
{
    if (!hWnd) return E_FAIL;

    ShowWindow(hWnd, SW_SHOW);
    UpdateWindow(hWnd);

    return S_OK;
}

CRC_API HRESULT CRC::CreateD3D11DeviceAndSwapChain
(
    const HWND& hWnd,
    Microsoft::WRL::ComPtr<ID3D11Device> &device, Microsoft::WRL::ComPtr<IDXGISwapChain> &swapChain
){
    // Setup swap chain
    DXGI_SWAP_CHAIN_DESC sd;
    ZeroMemory(&sd, sizeof(sd));
    sd.BufferCount = 2;
    sd.BufferDesc.Width = 0;
    sd.BufferDesc.Height = 0;
    sd.BufferDesc.Format = DXGI_FORMAT_R8G8B8A8_UNORM;
    sd.BufferDesc.RefreshRate.Numerator = 60;
    sd.BufferDesc.RefreshRate.Denominator = 1;
    sd.Flags = DXGI_SWAP_CHAIN_FLAG_ALLOW_MODE_SWITCH;
    sd.BufferUsage = DXGI_USAGE_RENDER_TARGET_OUTPUT;
    sd.OutputWindow = hWnd;
    sd.SampleDesc.Count = 1;
    sd.SampleDesc.Quality = 0;
    sd.Windowed = TRUE;
    sd.SwapEffect = DXGI_SWAP_EFFECT_FLIP_SEQUENTIAL;

    UINT createDeviceFlags = 0;
    D3D_FEATURE_LEVEL featureLevel;
    const D3D_FEATURE_LEVEL featureLevelArray[2] = { D3D_FEATURE_LEVEL_11_0, D3D_FEATURE_LEVEL_10_0, };

    HRESULT hr = D3D11CreateDeviceAndSwapChain
    (
        nullptr, D3D_DRIVER_TYPE_HARDWARE, nullptr, createDeviceFlags, featureLevelArray, 2, 
        D3D11_SDK_VERSION, &sd, &swapChain, &device, &featureLevel, nullptr
    );

    if (hr == DXGI_ERROR_UNSUPPORTED) // Try high-performance WARP software driver if hardware is not available.
    {
        hr = D3D11CreateDeviceAndSwapChain
        (
            nullptr, D3D_DRIVER_TYPE_WARP, nullptr, createDeviceFlags, featureLevelArray, 2, 
            D3D11_SDK_VERSION, &sd, &swapChain, &device, &featureLevel, nullptr
        );
    }

    return hr;
}

CRC_API HRESULT CRC::CreateCRCDeviceAndSwapChain
(
    Microsoft::WRL::ComPtr<ID3D11Device> &d3d11Device, Microsoft::WRL::ComPtr<IDXGISwapChain> &d3d11SwapChain, 
    std::unique_ptr<ICRCDevice> &crcDevice, std::unique_ptr<ICRCSwapChain> crcSwapChain
){
    return E_NOTIMPL;
}

UINT CRC::GetBytesPerPixel(const DXGI_FORMAT &format)
{
    switch (format)
    {
    case DXGI_FORMAT_R8G8B8A8_UNORM:
    case DXGI_FORMAT_D24_UNORM_S8_UINT:
        return 4;

    default:
        throw std::runtime_error("This DXGI_FORMAT is not supported by CuRendCore.");
    }
}

CRC_API void CRC::CheckCuda(hipError_t call)
{
    if (call != hipSuccess)
    {
        std::string err = "[CUDA ERROR] Code: " + std::to_string(call) + ", Reason: " + hipGetErrorString(call);
#ifndef NDEBUG
        CoutError(err);
#endif
        throw std::runtime_error(err);
    }
}

HRESULT CRC::RegisterCudaResources
(
    std::vector<hipGraphicsResource_t> &cudaResources, const hipGraphicsRegisterFlags &flags, 
    const UINT &bufferCount, IDXGISwapChain *d3d11SwapChain
){
    cudaResources.resize(bufferCount);
    std::vector<ID3D11Texture2D*> buffers(bufferCount);

    DXGI_SWAP_CHAIN_DESC desc;
    d3d11SwapChain->GetDesc(&desc);
    UINT bufferCountFromDesc = desc.BufferCount;

    for (UINT i = 0; i < bufferCountFromDesc; i++)
    {
        HRESULT hr = d3d11SwapChain->GetBuffer(i, __uuidof(ID3D11Texture2D), (void**)&buffers[i]);
        if (FAILED(hr)) return hr;

        hipError_t err = cudaGraphicsD3D11RegisterResource
        (
            &cudaResources[i], buffers[i], flags
        );
        if (err != hipSuccess) return E_FAIL;
    }

    for (int i = 0; i < bufferCountFromDesc; i++)
    {
        buffers[i]->Release();
    }

    return S_OK;
}

HRESULT CRC::RegisterCudaResource
(
    hipGraphicsResource_t &cudaResource, const hipGraphicsRegisterFlags &flags, 
    ID3D11Texture2D *d3d11Texture
){
    hipError_t err = cudaGraphicsD3D11RegisterResource(&cudaResource, d3d11Texture, flags);
    if (err != hipSuccess) return E_FAIL;

    return S_OK;
}

HRESULT CRC::UnregisterCudaResources(std::vector<hipGraphicsResource_t> &cudaResources)
{
    for (int i = 0; i < cudaResources.size(); ++i) 
    {
        hipError_t err = hipGraphicsUnregisterResource(cudaResources[i]);
        if (err != hipSuccess) return E_FAIL;
    }

    return S_OK;
}

HRESULT CRC::UnregisterCudaResource(hipGraphicsResource_t &cudaResource)
{
    hipError_t err = hipGraphicsUnregisterResource(cudaResource);
    if (err != hipSuccess) return E_FAIL;

    return S_OK;
}

HRESULT CRC::UnregisterCudaResourcesAtSwapChain
(
    std::vector<hipGraphicsResource_t> &cudaResources, 
    Microsoft::WRL::ComPtr<IDXGISwapChain> &d3d11SwapChain, UINT &frameIndex, const UINT& bufferCount
){
    for (int i = 0; i < cudaResources.size(); ++i) 
    {
        if (i == frameIndex) continue;

        hipError_t err = hipGraphicsUnregisterResource(cudaResources[i]);
        if (err != hipSuccess) return E_FAIL;
    }

    d3d11SwapChain->Present(0, 0);
    hipError_t err = hipGraphicsUnregisterResource(cudaResources[frameIndex]);
    if (err != hipSuccess) return E_FAIL;

    frameIndex = (frameIndex + 1) % bufferCount;

    return S_OK;
}

HRESULT CRC::MapCudaResource(hipGraphicsResource_t& cudaResource, hipStream_t stream)
{
    hipError_t err = hipGraphicsMapResources(1, &cudaResource, stream);
    if (err != hipSuccess) return E_FAIL;

    return S_OK;
}

HRESULT CRC::UnmapCudaResource(hipGraphicsResource_t& cudaResource, hipStream_t stream)
{
    hipError_t err = hipGraphicsUnmapResources(1, &cudaResource, stream);
    if (err != hipSuccess) return E_FAIL;

    return S_OK;
}

hipArray_t CRC::GetCudaMappedArray(hipGraphicsResource_t& cudaResource)
{
    hipArray_t hipArray;
    hipError_t err = hipGraphicsSubResourceGetMappedArray(&hipArray, cudaResource, 0, 0);
    if (err != hipSuccess) return nullptr;

    return hipArray;
}
