﻿#include "CRC_pch.h"
#include "CRC_funcs.cuh"

#include "CRC_memory.cuh"



// CRC_API HRESULT CRC::MallocMem(CRCCudaMem &mem, const std::size_t &size)
// {
//     mem.size_ = size;

//     if (mem.dPtr_)
//     {
//         throw std::runtime_error("Device memory already allocated.");
//         return E_FAIL;
//     }

//     CRC::CheckCuda(hipMalloc((void**)&mem.dPtr_, size));

//     if (CRC::As<CRCAccessEnabled>(&mem.hReadAccess) || CRC::As<CRCAccessEnabled>(&mem.hWriteAccess))
//     {
//         if (mem.hPtr_)
//         {
//             throw std::runtime_error("Host memory already allocated.");
//             return E_FAIL;
//         }
        
//         CRC::CheckCuda(hipHostMalloc((void**)&mem.hPtr_, size));
//     }
// }

// CRC_API HRESULT CRC::SetHostMem(CRCCudaMem &mem, const D3D11_SUBRESOURCE_DATA &initialData)
// {
//     if (!mem.hPtr_)
//     {
//         throw std::runtime_error("Host memory not allocated.");
//         return E_FAIL;
//     }

//     CRC::CheckCuda(hipMemcpy((void**)&mem.hPtr_, (void**)&initialData.pSysMem, mem.size_, hipMemcpyHostToHost));
// }

// CRC_API HRESULT CRC::SetDeviceMem(CRCCudaMem &mem, const D3D11_SUBRESOURCE_DATA &initialData)
// {
//     if (!mem.dPtr_)
//     {
//         throw std::runtime_error("Device memory not allocated.");
//         return E_FAIL;
//     }

//     CRC::CheckCuda(hipMemcpy((void**)&mem.dPtr_, (void**)&initialData.pSysMem, mem.size_, hipMemcpyHostToDevice));
// }

// CRC_API HRESULT CRC::SetMem(CRCCudaMem &mem, const D3D11_SUBRESOURCE_DATA &initialData)
// {
//     CRC::SetHostMem(mem, initialData);
//     CRC::SetDeviceMem(mem, initialData);
// }

// CRC_API HRESULT CRC::FreeHostMem(CRCCudaMem &mem)
// {
//     if (!mem.hPtr_)
//     {
//         throw std::runtime_error("Host memory not allocated.");
//         return E_FAIL;
//     }

//     CRC::CheckCuda(hipHostFree((void**)&mem.hPtr_));
// }

// CRC_API HRESULT CRC::FreeDeviceMem(CRCCudaMem &mem)
// {
//     if (!mem.dPtr_)
//     {
//         throw std::runtime_error("Device memory not allocated.");
//         return E_FAIL;
//     }

//     CRC::CheckCuda(hipFree((void**)&mem.dPtr_));
// }

// CRC_API HRESULT CRC::FreeMem(CRCCudaMem &mem)
// {
//     CRC::FreeHostMem(mem);
//     CRC::FreeDeviceMem(mem);
// }

CRC_API HRESULT CRC::MallocCudaMem
(
    CRCCudaMem &mem, const std::size_t &size, const UINT& pitch, const UINT& slicePitch
){
    if (mem.host || mem.device)
    {
        throw std::runtime_error("Memory already allocated.");
        return E_FAIL;
    }

    CRC::CheckCuda(hipHostMalloc((void**)&mem.host->Mem(), size));
    CRC::CheckCuda(hipMalloc((void**)&mem.host->Mem(), size));

    CRC::CheckCuda(hipMalloc((void**)&mem.device, sizeof(CRCMem)));

    return S_OK;
}

CRC_API HRESULT CRC::SetCudaMem(CRCCudaMem &mem, const void* sysMem)
{
    if (!mem.host || !mem.device)
    {
        throw std::runtime_error("Memory not allocated.");
        return E_FAIL;
    }

    CRC::CheckCuda(hipMemcpy
    (
        (void*)mem.host->Mem(), sysMem, 
        mem.host->Size(), hipMemcpyHostToHost
    ));

    CRC::CheckCuda(hipMemcpy
    (
        (void*)mem.device, mem.host, 
        sizeof(CRCMem), hipMemcpyHostToDevice
    ));

    return S_OK;
}

CRC_API HRESULT CRC::FreeCudaMem(CRCCudaMem &mem)
{
    if (!mem.host || !mem.device)
    {
        throw std::runtime_error("Memory not allocated.");
        return E_FAIL;
    }

    CRC::CheckCuda(hipFree((void**)&mem.host->Mem()));
    CRC::CheckCuda(hipHostFree((void**)&mem.host));

    CRC::CheckCuda(hipFree((void*)mem.device));
    return S_OK;
}
