﻿#include "CRC_pch.h"
#include "CRC_funcs.cuh"

#include "CRC_memory.cuh"

CRCHostMem::~CRCHostMem()
{
    if (ptr_) Free();
}

void CRCHostMem::Malloc(const UINT &byteWidth, const UINT &pitch, const UINT &slicePitch)
{
    if (ptr_)
    {
#ifndef NDEBUG
        CRC::CoutError("Memory already allocated.");
#endif
        throw std::runtime_error("Memory already allocated.");
    }

    byteWidth_ = byteWidth;
    pitch_ = pitch;
    slicePitch_ = slicePitch;

    CRC::CheckCuda(hipHostMalloc(&ptr_, byteWidth_));

#ifndef NDEBUG
    CRC::Cout
    (
        "Host memory allocated.", "\n", 
        "ByteWidth :", byteWidth_, "\n",
        "Pitch :", pitch_, "\n",
        "SlicePitch :", slicePitch_
    );
#endif
}

void CRCHostMem::Free()
{
    if (!ptr_)
    {
#ifndef NDEBUG
        CRC::CoutError("Memory not allocated.");
#endif
        throw std::runtime_error("Memory not allocated.");
    }

    byteWidth_ = 0;
    pitch_ = 0;
    slicePitch_ = 0;

    CRC::CheckCuda(hipHostFree(ptr_));
    ptr_ = nullptr;

#ifndef NDEBUG
    CRC::Cout("Host memory free.");
#endif
}

CRCDeviceMem::~CRCDeviceMem()
{
    if (ptr_) Free();
}

void CRCDeviceMem::Malloc(const UINT &byteWidth, const UINT &pitch, const UINT &slicePitch)
{
    if (ptr_)
    {
#ifndef NDEBUG
        CRC::CoutError("Memory already allocated.");
#endif
        throw std::runtime_error("Memory already allocated.");
    }

    byteWidth_ = byteWidth;
    pitch_ = pitch;
    slicePitch_ = slicePitch;

    CRC::CheckCuda(hipMalloc(&ptr_, byteWidth_));

#ifndef NDEBUG
    CRC::Cout
    (
        "Device memory allocated.", "\n", 
        "ByteWidth :", byteWidth_, "\n",
        "Pitch :", pitch_, "\n",
        "SlicePitch :", slicePitch_
    );
#endif
}

void CRCDeviceMem::Free()
{
    if (!ptr_)
    {
#ifndef NDEBUG
        CRC::CoutError("Memory not allocated.");
#endif
        throw std::runtime_error("Memory not allocated.");
    }

    byteWidth_ = 0;
    pitch_ = 0;
    slicePitch_ = 0;

    CRC::CheckCuda(hipFree(ptr_));
    ptr_ = nullptr;

#ifndef NDEBUG
    CRC::Cout("Device memory free.");
#endif
}
