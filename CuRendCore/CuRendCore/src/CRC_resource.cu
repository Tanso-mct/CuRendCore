#include "CRC_pch.h"
#include "CRC_funcs.cuh"

#include "CRC_resource.cuh"

CRC_API void CRC::MallocCudaMem(CRC_CUDA_MEMORY &mem, const std::size_t &size)
{
    mem.size_ = size;
    CRC::CheckCuda(hipMalloc(&mem.ptr_, size));
}

CRC_API void CRC::SetCudaMem(CRC_CUDA_MEMORY &mem, const D3D11_SUBRESOURCE_DATA &initialData)
{
    CRC::CheckCuda(hipMemcpy(mem.ptr_, initialData.pSysMem, mem.size_, hipMemcpyHostToDevice));
}

CRC_API void CRC::FreeCudaMem(CRC_CUDA_MEMORY &mem)
{
    CRC::CheckCuda(hipFree(mem.ptr_));
}
