﻿#include "CRC_pch.h"
#include "CRC_funcs.h"

#include "CRC_rtv.cuh"

std::unique_ptr<ICRCResource> &CRCRenderTargetView::GetResource()
{
    return resource_;
}

std::unique_ptr<ICRCResource> &CRCID3D11RenderTargetView::GetResource()
{
    return emptyResource_;
}
