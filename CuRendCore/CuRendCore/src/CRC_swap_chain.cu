﻿#include "CRC_pch.h"
#include "CRC_funcs.cuh"

#include "CRC_swap_chain.cuh"

std::unique_ptr<ICRCContainable> CRCSwapChainFactoryL0_0::Create(IDESC &desc) const
{
    CRC_SWAP_CHAIN_DESC* swapChainDesc = CRC::As<CRC_SWAP_CHAIN_DESC>(&desc);
    if (!swapChainDesc) return nullptr;

    std::unique_ptr<CRCSwapChain> swapChain = std::make_unique<CRCSwapChain>
    (
        swapChainDesc->GetD3D11SwapChain(),
        swapChainDesc->BufferCount(),
        swapChainDesc->BufferUsage(),
        swapChainDesc->RefreshRate(),
        swapChainDesc->SwapEffect()
    );

    return swapChain;
}

CRCSwapChain::CRCSwapChain
(
    Microsoft::WRL::ComPtr<IDXGISwapChain> &d3d11SwapChain, 
    UINT bufferCount, DXGI_USAGE bufferUsage, DXGI_RATIONAL refreshRate, DXGI_SWAP_EFFECT swapEffect
) 
: d3d11SwapChain_(d3d11SwapChain)
, bufferCount_(bufferCount), bufferUsage_(bufferUsage), refreshRate_(refreshRate), swapEffect_(swapEffect)
{
    HRESULT hr = CRC::RegisterCudaResources
    (
        cudaResources_, hipGraphicsRegisterFlagsNone,
        bufferCount, d3d11SwapChain_.Get()
    );
    if (FAILED(hr))
    {
#ifndef NDEBUG
        CRC::CoutError("Failed to create CRCSwapChain by registering CUDA resources.");
#endif
        throw std::runtime_error("Failed to create CRCSwapChain by registering CUDA resources.");
    }

    hr = CRC::MapCudaResource(cudaResources_[frameIndex_]);
    if (FAILED(hr))
    {
#ifndef NDEBUG
        CRC::CoutError("Failed to create CRCSwapChain by mapping CUDA resources.");
#endif
        throw std::runtime_error("Failed to create CRCSwapChain by mapping CUDA resources.");
    }

    hipArray_t backBufferArray = CRC::GetCudaMappedArray(cudaResources_[frameIndex_]);
    if (!backBufferArray)
    {
#ifndef NDEBUG
        CRC::CoutError("Failed to create CRCSwapChain by getting CUDA mapped array.");
#endif
        throw std::runtime_error("Failed to create CRCSwapChain by getting CUDA mapped array.");
    }

    backBuffer_ = std::make_unique<CRCTexture2D>();
    backBuffer_->GetPtr() = (void*)backBufferArray;
}

CRCSwapChain::~CRCSwapChain()
{
    CRC::UnmapCudaResource(cudaResources_[frameIndex_]);
    CRC::UnregisterCudaResourcesAtSwapChain(cudaResources_, d3d11SwapChain_, frameIndex_, bufferCount_);

    backBuffer_->GetPtr() = nullptr;
    backBuffer_.reset();
}

HRESULT CRCSwapChain::GetBuffer(UINT buffer, ICRCTexture2D *&texture)
{
    texture = backBuffer_.get();
    if (!texture) return E_INVALIDARG;
    return S_OK;
}

HRESULT CRCSwapChain::ResizeBuffers
(
    UINT bufferCount, UINT width, UINT height, DXGI_FORMAT newFormat, UINT swapChainFlags
){
    HRESULT hr = S_OK;
    hr = CRC::UnmapCudaResource(cudaResources_[frameIndex_]);
    if (FAILED(hr)) return hr;

    hr = CRC::UnregisterCudaResourcesAtSwapChain(cudaResources_, d3d11SwapChain_, frameIndex_, bufferCount_);
    if (FAILED(hr)) return hr;

    hr = d3d11SwapChain_->ResizeBuffers(bufferCount, width, height, newFormat, swapChainFlags);
    if (FAILED(hr)) return hr;

    frameIndex_ = 0;

    hr = CRC::RegisterCudaResources
    (
        cudaResources_, hipGraphicsRegisterFlagsNone, 
        bufferCount, d3d11SwapChain_.Get()
    );
    if (FAILED(hr)) return hr;

    hr = CRC::MapCudaResource(cudaResources_[frameIndex_]);
    if (FAILED(hr)) return hr;

    hipArray_t backBufferArray = CRC::GetCudaMappedArray(cudaResources_[frameIndex_]);
    if (!backBufferArray) return E_FAIL;

    backBuffer_->GetPtr() = (void*)backBufferArray;

    return S_OK;
}

HRESULT CRCSwapChain::Present(UINT syncInterval, UINT flags)
{
    HRESULT hr = CRC::UnmapCudaResource(cudaResources_[frameIndex_]);
    if (FAILED(hr)) return hr;

    hr = d3d11SwapChain_->Present(syncInterval, flags);
    if (FAILED(hr)) return hr;

    frameIndex_ = (frameIndex_ + 1) % bufferCount_;

    hr = CRC::MapCudaResource(cudaResources_[frameIndex_]);
    if (FAILED(hr)) return hr;

    hipArray_t backBufferArray = CRC::GetCudaMappedArray(cudaResources_[frameIndex_]);
    if (!backBufferArray) return E_FAIL;

    backBuffer_->GetPtr() = (void*)backBufferArray;

    return S_OK;
}

std::unique_ptr<ICRCContainable> CRCIDXGISwapChainFactoryL0_0::Create(IDESC &desc) const
{
    CRC_SWAP_CHAIN_DESC* swapChainDesc = CRC::As<CRC_SWAP_CHAIN_DESC>(&desc);
    if (!swapChainDesc) return nullptr;

    std::unique_ptr<CRCIDXGISwapChain> swapChain = std::make_unique<CRCIDXGISwapChain>
    (
        swapChainDesc->GetD3D11SwapChain()
    );

    return swapChain;
}

HRESULT CRCIDXGISwapChain::GetBuffer(UINT buffer, ICRCTexture2D *&texture)
{
    CRCID3D11Texture2D* backBuffer = CRC::As<CRCID3D11Texture2D>(texture);
    if (!backBuffer) return E_INVALIDARG;

    d3d11SwapChain_->GetBuffer(buffer, __uuidof(ID3D11Texture2D), &backBuffer->Get());
}

HRESULT CRCIDXGISwapChain::ResizeBuffers
(
    UINT bufferCount, UINT width, UINT height, DXGI_FORMAT newFormat, UINT swapChainFlags
){
    return d3d11SwapChain_->ResizeBuffers(bufferCount, width, height, newFormat, swapChainFlags);
}

HRESULT CRCIDXGISwapChain::Present(UINT syncInterval, UINT flags)
{
    return d3d11SwapChain_->Present(syncInterval, flags);
}
